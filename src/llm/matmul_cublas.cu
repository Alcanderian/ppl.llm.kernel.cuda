#include "hip/hip_runtime.h"
#include "cudakernel/llm/matmul_cublas.h"
#include <stdio.h>
#if __CUDACC_VER_MAJOR__ >= 9 && !defined(_WIN64)
#include <mutex>
static bool g_is_less_volta_deivce_ = false;
static std::once_flag is_less_volta_deivce_onceflag;
#endif

static void isLessVolta() {
#if __CUDACC_VER_MAJOR__ >= 9 && !defined(_WIN64)
    int dev_id_;
    hipError_t result_  = hipCtxGetDevice(&dev_id_);
    if (result_ != hipSuccess ) { g_is_less_volta_deivce_ = false; }
    hipDeviceProp_t deviceProp;
    auto err = hipGetDeviceProperties(&deviceProp, dev_id_);
    if (err != hipSuccess) { g_is_less_volta_deivce_ = false; }
    if (deviceProp.major < 7) {
        g_is_less_volta_deivce_  = true;
    } else {
        g_is_less_volta_deivce_ = false;
    }
#endif
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        throw std::logic_error("cuBLAS API failed");
    }
}

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        throw std::logic_error("cuda API failed");
    }
}

uint64_t PPLCUDAMatMulCublasGetRuntimeBufSize(
        const ppl::common::TensorShape* input_shape,
        const ppl::common::TensorShape* weight_shape) {
    auto dim_count0 = input_shape->GetDimCount();
    auto dim_count1 = weight_shape->GetDimCount();
    int m_id = dim_count0 - 2;
    uint64_t batch = 1;
    if (dim_count1 == 2){
        batch = 1;
    } else {
        for (int i = 0; i < m_id; i++) 
            batch *= input_shape->GetDim(i);
    }
    return 1024 * 1024 * 4 * batch;
}

double PPLCUDAMatMulCublasSelectKernel(
            const hipStream_t stream,
            const hipblasLtHandle_t& ltHandle,
            const GemmKernelParam &param, 
            const ppl::common::TensorShape* input_shape,
            const void* input,
            const ppl::common::TensorShape* weight_shape,
            const void* weight,
            const ppl::common::TensorShape* output_shape,
            void* output,
            const ppl::common::TensorShape* bias_shape,
            const void* bias,
            const int64_t batch,
            void* workspace, 
            size_t workspaceSize,
            hipblasLtMatmulAlgo_t& algo) {
    auto dim_count0 = input_shape->GetDimCount();
    auto dim_count1 = weight_shape->GetDimCount();
    int m_id = dim_count0 - 2;
    int k_id = dim_count0 - 1;
    int n_id = dim_count1 - 1;
    if (param.transA) {
        m_id = dim_count0 - 1;
        k_id = dim_count0 - 2;
    }
    if (param.transB) {
        n_id = dim_count1 - 2;
    }
    int64_t M = input_shape->GetDim(m_id);
    if (dim_count1 == 2 && dim_count0 > 2){ // matmul case
        for (int i = 0; i < m_id; i++){
            M *= input_shape->GetDim(i);
        }
    }
    int64_t K     = input_shape->GetDim(k_id);
    int64_t N     = weight_shape->GetDim(n_id);
    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr, Ddesc = nullptr;

    hipblasOperation_t transa = param.transA == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = param.transB == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    float alpha = 1;
    float beta = 0.f;
    int64_t lda = param.transA ? M : K;
    int64_t ldb = param.transB ? K : N;
    int64_t ldc = N;
    hipDataType dt = HIP_R_32F;
    if (input_shape->GetDataType() == ppl::common::DATATYPE_FLOAT16) {
        dt = HIP_R_16F;
    }

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 >= 11000
    hipblasComputeType_t ct = HIPBLAS_COMPUTE_32F_FAST_TF32;
    if (input_shape->GetDataType() == ppl::common::DATATYPE_FLOAT16) {
        ct = HIPBLAS_COMPUTE_32F;
    }
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, ct, dt));
#elif __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 > 10000
    hipDataType ct = dt;
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, ct));
#endif
    // exchange A & B
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transb, sizeof(transb)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transa, sizeof(transa)));
    if(batch == 1) {
        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, dt, transa == HIPBLAS_OP_N ? K : M, transa == HIPBLAS_OP_N ? M : K, lda));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, dt, transb == HIPBLAS_OP_N ? N : K, transb == HIPBLAS_OP_N ? K : N, ldb));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, dt, N, M, ldc));
    } else {
        int64_t stridea = M * K;
        int64_t strideb = K * N;
        int64_t stridec = M * N;
        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, dt, transa == HIPBLAS_OP_N ? K : M, transa == HIPBLAS_OP_N ? M : K, lda));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea)));

        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, dt, transb == HIPBLAS_OP_N ? N : K, transb == HIPBLAS_OP_N ? K : N, ldb));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb)));

        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, dt, N, M, ldc));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec)));
    }
    // **************** Request Algos **************
    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    hipblasLtMatmulPreference_t preference = nullptr;
    constexpr int requested_algo = 8;
    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult[requested_algo] = { 0 };
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));
    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Bdesc, Adesc, Cdesc, Cdesc, preference, requested_algo, heuristicResult, &returnedResults));
    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    // run and evaluate timing
    constexpr int repeatAlgoCheck = 5;
    int bestAlgoIdx = 0;
    float time = 0;
    float bestAlgoTime = 0;
    hipEvent_t startEvent, stopEvent;
    checkCudaStatus(hipEventCreate(&startEvent));
    checkCudaStatus(hipEventCreate(&stopEvent));
    for (int algoIdx = 0; algoIdx < returnedResults; algoIdx++) {
        checkCudaStatus(hipEventRecord(startEvent, stream));
        for (int checkIdx = 0; checkIdx < repeatAlgoCheck; checkIdx++) {
            checkCublasStatus(hipblasLtMatmul(ltHandle,
                                        operationDesc,
                                        (const void*)(&alpha),
                                        weight,
                                        Bdesc,
                                        input,
                                        Adesc,
                                        (const void*)(&beta),
                                        output,
                                        Cdesc,
                                        output,
                                        Cdesc,
                                        &heuristicResult[algoIdx].algo,
                                        workspace,
                                        workspaceSize,
                                        stream));
        }
        checkCudaStatus(hipEventRecord(stopEvent, stream));
        checkCudaStatus(hipEventSynchronize(stopEvent));
        checkCudaStatus(hipEventElapsedTime(&time, startEvent, stopEvent));
        time /= repeatAlgoCheck;

        if (algoIdx == 0 || time < bestAlgoTime) {
            bestAlgoTime = time;
            bestAlgoIdx = algoIdx;
        }
    }
    memcpy(&algo, &heuristicResult[bestAlgoIdx].algo, sizeof(algo));

    if (startEvent) checkCudaStatus(hipEventDestroy(startEvent));
    if (stopEvent) checkCudaStatus(hipEventDestroy(stopEvent));
    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
    return bestAlgoTime;
}

template <typename T>
__global__ void ppl_matmul_add_bias(int32_t num_elems, int32_t bias_len, T* output, const T* bias) {
    int b_idx= blockIdx.y;
    int index = threadIdx.x + blockIdx.x * blockDim.x; 
    if (index >= num_elems) return;
    int32_t bias_offset = b_idx * bias_len + index % bias_len;
    int32_t output_offset = b_idx * num_elems + index;
    output[output_offset] += bias[bias_offset];
}

ppl::common::RetCode PPLCUDAMatMulCublasForwardImp(
        const hipStream_t stream,
        const hipblasLtHandle_t& ltHandle,
        const GemmKernelParam &param,
        const ppl::common::TensorShape* input_shape,
        const void* input,
        const ppl::common::TensorShape* weight_shape,
        const void* weight,
        const ppl::common::TensorShape* output_shape,
        void* output,
        const ppl::common::TensorShape* bias_shape,
        const void* bias,
        const int64_t batch,
        void* workspace, 
        size_t workspaceSize,
        bool use_heuristic,
        hipblasLtMatmulAlgo_t algo) {
#if __CUDACC_VER_MAJOR__ >= 9 && !defined(_WIN64)
    std::call_once(is_less_volta_deivce_onceflag, isLessVolta);
    if (g_is_less_volta_deivce_) {
        use_heuristic = false; // cublas matmul is tested crash on GTX1060, but success on volta, turing, ampere
    }
#endif
    
    auto dim_count0 = input_shape->GetDimCount();
    auto dim_count1 = weight_shape->GetDimCount();
    int m_id = dim_count0 - 2;
    int k_id = dim_count0 - 1;
    int n_id = dim_count1 - 1;
    if (param.transA) {
        m_id = dim_count0 - 1;
        k_id = dim_count0 - 2;
    }
    if (param.transB) {
        n_id = dim_count1 - 2;
    }
    int64_t M = input_shape->GetDim(m_id);
    if (dim_count1 == 2 && dim_count0 > 2){ // matmul case
        for (int i = 0; i < m_id; i++){
            M *= input_shape->GetDim(i);
        }
    }
    int64_t K     = input_shape->GetDim(k_id);
    int64_t N     = weight_shape->GetDim(n_id);
    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr, Ddesc = nullptr;
    hipblasLtMatmulPreference_t preference = nullptr;

    hipblasOperation_t transa = param.transA == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = param.transB == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    float alpha = param.alpha;
    float beta = 0.f;
    int lda = param.transA ? M : K;
    int ldb = param.transB ? K : N;
    int ldc = N;
    hipDataType dt = HIP_R_32F;
    using Dtype = float;
    if (input_shape->GetDataType() == ppl::common::DATATYPE_FLOAT16) {
        dt = HIP_R_16F;
        using Dtype = half;
    }

#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 >= 11000
    hipblasComputeType_t ct = HIPBLAS_COMPUTE_32F_FAST_TF32;
    if (input_shape->GetDataType() == ppl::common::DATATYPE_FLOAT16) {
        ct = HIPBLAS_COMPUTE_32F;
    }
    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, ct, dt));
#elif __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 > 10000
    hipDataType ct = dt;
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, ct));
#endif

    bias = reinterpret_cast<const Dtype*>(bias);
    // exchange A & B
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transb, sizeof(transb)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transa, sizeof(transa)));
#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 >= 11000
    if (bias != nullptr) {
        hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(Dtype*)));
    }
#endif
    if(batch == 1) {
        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, dt, transa == HIPBLAS_OP_N ? K : M, transa == HIPBLAS_OP_N ? M : K, lda));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, dt, transb == HIPBLAS_OP_N ? N : K, transb == HIPBLAS_OP_N ? K : N, ldb));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, dt, N, M, ldc));
    } else {
        int64_t stridea = M * K;
        int64_t strideb = K * N;
        int64_t stridec = M * N;
        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, dt, transa == HIPBLAS_OP_N ? K : M, transa == HIPBLAS_OP_N ? M : K, lda));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea)));

        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, dt, transb == HIPBLAS_OP_N ? N : K, transb == HIPBLAS_OP_N ? K : N, ldb));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb)));

        checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, dt, N, M, ldc));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(batch)));
        checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec)));
    }

    #if 1
    if (use_heuristic) {
        hipblasLtMatmulHeuristicResult_t heurResult;
        hipblasStatus_t algoStatus = cublasLtMatmulAlgoCheck(ltHandle,
                                operationDesc,
                                Bdesc,
                                Adesc,
                                Cdesc,
                                Cdesc,
                                &algo, 
                                &heurResult);  
        if (algoStatus != HIPBLAS_STATUS_SUCCESS) {
            use_heuristic = false;
        }
    }
    #endif

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                    operationDesc,
                                    (const void*)(&alpha),
                                    weight,
                                    Bdesc,
                                    input,
                                    Adesc,
                                    (const void*)(&beta),
                                    output,
                                    Cdesc,
                                    output,
                                    Cdesc,
                                    use_heuristic ? &algo : nullptr,
                                    workspace,
                                    workspaceSize,
                                    stream));

// cuda 10.1&10.2 need tackle bias alone
#if __CUDACC_VER_MAJOR__ * 1000 + __CUDACC_VER_MINOR__ * 10 < 11000
    if (bias != nullptr) {
        int64_t num_elems = M * N;
        int64_t block_size = 128;
        int64_t blocks = (num_elems + block_size - 1) / block_size;
        dim3 grid_size(blocks, batch, 1);
        ppl_matmul_add_bias<Dtype><<<grid_size, block_size, 0, stream>>>(num_elems, N, (Dtype*)output, (const Dtype*)bias);
    }
#endif

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
    
    return ppl::common::RC_SUCCESS;
}